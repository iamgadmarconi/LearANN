
#include <hip/hip_runtime.h>
// math.cu
extern "C" {
    __global__ void matrixMulKernel(float* A, float* B, float* C, int N) {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        if (row < N && col < N) {
            float value = 0;
            for (int k = 0; k < N; ++k) {
                value += A[row * N + k] * B[k * N + col];
            }
            C[row * N + col] = value;
        }
    }

    __global__ void dotProductKernel(float* A, float* B, float* C, int N) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < N) {
            C[idx] = A[idx] * B[idx];
        }
    }
}